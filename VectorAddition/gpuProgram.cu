#include "hip/hip_runtime.h"
// Created By: Sandeep Katragadda
// https://github.com/ksandeep4u/CUDA-examples

#pragma once
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gpuProgram.cuh"
#include<iostream>
#include<string>

using namespace std;

// CUDA kernel
__global__ void add_kernel(int n, float* x, float* y) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;
	for (int i = idx; i < n; i += stride)
		y[i] = x[i] + y[i];
}

void gpuProgram(int n) {
	cout << "GPU program!" << endl;
	int v = 0;
	hipRuntimeGetVersion(&v);
	string version = to_string(v / 1000) + "." + to_string(v % 1000);
	cout << "CUDA run time version: " << version << endl;
	hipDriverGetVersion(&v);
	version = to_string(v / 1000) + "." + to_string(v % 1000);
	cout << "CUDA driver version: " << version << endl;

	int N = n;

	// initialization
	float* x;
	float* y;

	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1;
		y[i] = 2;
	}

	// call to kernel
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add_kernel << <numBlocks, blockSize >> > (N, x, y);
	hipDeviceSynchronize();

	// display result
	//for (int i = 0; i < N; i++)
	//	cout << y[i] << endl;

	//check for error
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	cout << "Max error = " << maxError << endl;

	hipFree(x);
	hipFree(y);
}
